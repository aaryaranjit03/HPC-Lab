/* demo program for 2D grid with 2D blocks */

#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

__global__ void hello() {

//Global_Block_ID =blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;

    int BlockID, threadID;
    //BlockID=(gridDim.x * blockIdx.y + blockIdx.x);
		int Global_Block_ID =blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;
    printf("Global BID : %d |I am thread (%d, %d, %d) of block (%d, %d, %d) in the grid\n",
           Global_Block_ID,threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z );

}


void printDims(dim3 gridDim, dim3 blockDim) {
    printf("Grid Dimensions : {%d, %d, %d} blocks. \n",
    gridDim.x, gridDim.y, gridDim.z);

    printf("Block Dimensions : {%d, %d, %d} threads.\n",
    blockDim.x, blockDim.y, blockDim.z);
}

int main(int argc, char **argv) {


    dim3 gridDim(2,2);     // 2 blocks in x and y direction, z default to 1
    dim3 blockDim(2,2);  // 4 threads per block: 2 in x direction, 2 in y


    printDims(gridDim, blockDim);

    printf("From each thread:\n");
    hello<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();      // need for printfs in kernel to flush

    return 0;
}
